#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void HelloWorld()
{
    printf("Hello world, %d, %d\n", blockIdx.x, threadIdx.x); 
}

int main(){

    HelloWorld <<< 2, 5 >>>(); // Первое число -- количество блоков, Второе -- количество потоков

    hipDeviceSynchronize();

    return 0;
}